/*********************************************************************
 * Copyright © 2011-2012,
 * Marwan Abdellah: <abdellah.marwan@gmail.com>
 *
 * This library is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation.

 * This library is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.

 * You should have received a copy of the GNU Lesser General Public
 * License along with this library; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston,
 * MA 02110-1301, USA.
 ********************************************************************/

#ifndef _FILL_1D_ARRAY_RND_IMPL_CU_
#define _FILL_1D_ARRAY_RND_IMPL_CU_

#include "cu_Globals.h"
#include "Timers/BoostTimers.h"

#include "Kernels/Fill_1D_Array_RND_Kernel.cu"

template <typename T>
extern
void cu_Fill_1D_Array_RND_Impl
(dim3 cuBlock, dim3 cuGrid, T *devArray, int N,
 cuProfile* profile)
{
    // Create CUDA timer
    cutCreateTimer(&(profile->kernelTime));

    // Reset CUDA timer
    cutResetTimer(profile->kernelTime);

    // Start CUDA timer
    cutStartTimer(profile->kernelTime);

    // Execute the kernel
    Fill_1D_Array_RND_Kernel <<< cuGrid, cuBlock >>> (devArray, N);

    // Stop CUDA timer
    cutStopTimer(profile->kernelTime);

    // Calculate kernel execution time
    profile->kernelDuration = cutGetTimerValue(profile->kernelTime);

    // Check successfull execution of the kernel
    profile->kernelExecErr = hipPeekAtLastError();
}

template
void cu_Fill_1D_Array_RND_Impl <char>
(dim3 cuBlock, dim3 cuGrid,
char *devArray, int N,
cuProfile* profile);

template
void cu_Fill_1D_Array_RND_Impl <unsigned char>
(dim3 cuBlock, dim3 cuGrid,
unsigned char *devArray, int N,
cuProfile* profile);

template
void cu_Fill_1D_Array_RND_Impl <int>
(dim3 cuBlock, dim3 cuGrid,
int *devArray, int N,
cuProfile* profile);

template
void cu_Fill_1D_Array_RND_Impl <unsigned int>
(dim3 cuBlock, dim3 cuGrid,
unsigned int *devArray, int N,
cuProfile* profile);

template
void cu_Fill_1D_Array_RND_Impl <float>
(dim3 cuBlock, dim3 cuGrid,
float *devArray, int N,
cuProfile* profile);

template
void cu_Fill_1D_Array_RND_Impl <double>
(dim3 cuBlock, dim3 cuGrid,
double *devArray, int N,
cuProfile* profile);

#endif // _FILL_1D_ARRAY_RND_IMPL_CU_
