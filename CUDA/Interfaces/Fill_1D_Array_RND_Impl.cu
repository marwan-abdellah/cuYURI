/*********************************************************************
 * Copyright © 2011-2012,
 * Marwan Abdellah: <abdellah.marwan@gmail.com>
 *
 * This library is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation.

 * This library is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.

 * You should have received a copy of the GNU Lesser General Public
 * License along with this library; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston,
 * MA 02110-1301, USA.
 ********************************************************************/

#ifndef _FILL_1D_ARRAY_RND_IMPL_CU_
#define _FILL_1D_ARRAY_RND_IMPL_CU_

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "cuGlobals.h"
#include "Timers/Boost.h"

#include "Kernels/Fill_1D_Array_RND_Kernel.cu"

template <typename T>
extern
void cu_Fill_1D_Array_RND_Impl
(dim3 cuBlock, dim3 cuGrid, T *devArray, int N,
 cuProfile* profile)
{
    // Create CUDA timer
    cutCreateTimer(&(profile->kernelTime));

    // Reset CUDA timer
    cutResetTimer(profile->kernelTime);

    // Start CUDA timer
    cutStartTimer(profile->kernelTime);

    // Random number generation code on the GPU
    hiprandState *stateDeviceMatrix;
    hipMalloc(&stateDeviceMatrix, cuBlock.x * cuGrid.x);

    // Init the random value
    initSeed_Kernel <<< cuGrid.x, cuBlock.x >>> (stateDeviceMatrix);

    // Execute the kernel
    Fill_1D_Array_RND_Kernel <<< cuGrid, cuBlock >>> (devArray, N, stateDeviceMatrix);

    // Free the state matrix on the device
    hipFree(stateDeviceMatrix);

    // Stop CUDA timer
    cutStopTimer(profile->kernelTime);

    // Calculate kernel execution time
    profile->kernelDuration = cutGetTimerValue(profile->kernelTime);

    // Check successfull execution of the kernel
    profile->kernelExecErr = hipPeekAtLastError();
}

template
void cu_Fill_1D_Array_RND_Impl <char>
(dim3 cuBlock, dim3 cuGrid,
char *devArray, int N,
cuProfile* profile);

template
void cu_Fill_1D_Array_RND_Impl <unsigned char>
(dim3 cuBlock, dim3 cuGrid,
unsigned char *devArray, int N,
cuProfile* profile);

template
void cu_Fill_1D_Array_RND_Impl <int>
(dim3 cuBlock, dim3 cuGrid,
int *devArray, int N,
cuProfile* profile);

template
void cu_Fill_1D_Array_RND_Impl <unsigned int>
(dim3 cuBlock, dim3 cuGrid,
unsigned int *devArray, int N,
cuProfile* profile);

template
void cu_Fill_1D_Array_RND_Impl <float>
(dim3 cuBlock, dim3 cuGrid,
float *devArray, int N,
cuProfile* profile);

template
void cu_Fill_1D_Array_RND_Impl <double>
(dim3 cuBlock, dim3 cuGrid,
double *devArray, int N,
cuProfile* profile);

#endif // _FILL_1D_ARRAY_RND_IMPL_CU_
