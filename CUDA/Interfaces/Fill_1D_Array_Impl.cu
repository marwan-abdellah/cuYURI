/*********************************************************************
 * Copyright © 2011-2012,
 * Marwan Abdellah: <abdellah.marwan@gmail.com>
 *
 * This library is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation.

 * This library is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.

 * You should have received a copy of the GNU Lesser General Public
 * License along with this library; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston,
 * MA 02110-1301, USA.
 ********************************************************************/

#ifndef _FILL_1D_ARRAY_IMPL_CU_
#define _FILL_1D_ARRAY_IMPL_CU_

#include "cuGlobals.h"
#include "Timers/Boost.h"

#include "Kernels/Fill_1D_Array_Kernel.cu"

template <typename T>
extern
void cu_Fill_1D_Array_Impl
(dim3 cuBlock, dim3 cuGrid, T *devArray, int N,
 int fillOrder, cuProfile* profile)
{
    // Create CUDA timer
    cutCreateTimer(&(profile->kernelTime));

    // Reset CUDA timer
    cutResetTimer(profile->kernelTime);

    // Start CUDA timer
    cutStartTimer(profile->kernelTime);

    // Execute the kernel
    Fill_1D_Array_Kernel <<< cuGrid, cuBlock >>> (devArray, N, fillOrder);

    // Stop CUDA timer
    cutStopTimer(profile->kernelTime);

    // Calculate kernel execution time
    profile->kernelDuration = cutGetTimerValue(profile->kernelTime);

    // Check successfull execution of the kernel
    profile->kernelExecErr = hipPeekAtLastError();
}

template
void cu_Fill_1D_Array_Impl <char>
(dim3 cuBlock, dim3 cuGrid,
char *devArray, int N,
int fillOrder, cuProfile* profile);

template
void cu_Fill_1D_Array_Impl <unsigned char>
(dim3 cuBlock, dim3 cuGrid,
unsigned char *devArray, int N,
int fillOrder, cuProfile* profile);

template
void cu_Fill_1D_Array_Impl <int>
(dim3 cuBlock, dim3 cuGrid,
int *devArray, int N,
int fillOrder, cuProfile* profile);

template
void cu_Fill_1D_Array_Impl <unsigned int>
(dim3 cuBlock, dim3 cuGrid,
unsigned int *devArray, int N,
int fillOrder, cuProfile* profile);

template
void cu_Fill_1D_Array_Impl <float>
(dim3 cuBlock, dim3 cuGrid,
float *devArray, int N,
int fillOrder, cuProfile* profile);

template
void cu_Fill_1D_Array_Impl <double>
(dim3 cuBlock, dim3 cuGrid,
double *devArray, int N,
int fillOrder, cuProfile* profile);

#endif // _FILL_1D_ARRAY_IMPL_CU_
