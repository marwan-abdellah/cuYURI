/*********************************************************************
 * Copyrights (c) Marwan Abdellah. All rights reserved.
 * This code is part of my Master's Thesis Project entitled "High
 * Performance Fourier Volume Rendering on Graphics Processing Units
 * (GPUs)" and submitted to the Systems & Biomedical Engineering
 * Department, Faculty of Engineering, Cairo University.
 * Please, don't use or distribute without authors' permission.

 * File         : Volume
 * Author(s)    : Marwan Abdellah <abdellah.marwan@gmail.com>
 * Created      : April 2011
 * Description  :
 * Note(s)      :
 *********************************************************************/
 
#ifndef CU_EXTERNS_CU_
#define CU_EXTERNS_CU_

#include "cuGlobals.h"

#include "cuCopyArray.cu"
#include "cuFillArray.cu"

#include "FFT/Real/cuFFTShift_2D_Real.cu"
#include "FFT/Real/cuFFTShift_3D_Real.cu"
#include "FFT/Complex/cuFFTShift_2D_Complex.cu"
#include "FFT/Complex/cuFFTShift_3D_Complex.cu"
#include "Timers/BoostTimers.h"





template <typename T>
extern
void cuFillArray(dim3 cuBlock, dim3 cuGrid, int *devArray, int nX);

template <typename T>
void cuFillArray(dim3 cuBlock, dim3 cuGrid, T *devArray, int nX)
{
    FillArray_Kernel <T> <<< cuGrid, cuBlock >>> (devArray, nX);

}

extern
void cuFillArray(dim3 cuBlock, dim3 cuGrid, int *devArray, int nX)
{
    FillArray_Kernel <<< cuGrid, cuBlock >>> (devArray, nX);
}














extern 
void cuFFTShift_2D( dim3 cuBlock, dim3 cuGrid,
                    float* devArrayOutput, float* devArrayInput, 
                    int nX, 
                    cudaProfile* cuProfile)
{   
    cutCreateTimer(&(cuProfile->kernelTime));
    cutResetTimer(cuProfile->kernelTime);
    cutStartTimer(cuProfile->kernelTime);
     
    fftShift_2D_Kernel <<< cuGrid, cuBlock >>> (devArrayOutput, devArrayInput, nX);
    hipDeviceSynchronize(); 
    
    cutStopTimer(cuProfile->kernelTime);
    
    cuProfile->kernelDuration = cutGetTimerValue(cuProfile->kernelTime);
    cuProfile->kernelExecErr = hipPeekAtLastError();
}

extern 
void cuFFTShift_2D_Double( dim3 cuBlock, dim3 cuGrid, 
                           double* devArrayOutput, double* devArrayInput, 
                           int nX, 
                           cudaProfile* cuProfile)
{
    cutCreateTimer(&(cuProfile->kernelTime));
    cutResetTimer(cuProfile->kernelTime);
    cutStartTimer(cuProfile->kernelTime);
    
    fftShift_2D_Double_Kernel <<< cuGrid, cuBlock>>> (devArrayOutput, devArrayInput, nX); 
    hipDeviceSynchronize(); 

    cutStopTimer(cuProfile->kernelTime);
    
    cuProfile->kernelDuration = cutGetTimerValue(cuProfile->kernelTime);
    cuProfile->kernelExecErr = hipPeekAtLastError();
}

extern 
void cuFFTShift_2D_Complex( dim3 cuBlock, dim3 cuGrid, 
                            hipfftComplex* devArrayOutput, hipfftComplex* devArrayInput, 
                            int nX,
                            cudaProfile* cuProfile)
{
    cutCreateTimer(&(cuProfile->kernelTime));
    cutResetTimer(cuProfile->kernelTime);
    cutStartTimer(cuProfile->kernelTime);
    
    fftShift_2D_Complex_Kernel <<< cuGrid, cuBlock>>> (devArrayOutput, devArrayInput, nX);
    hipDeviceSynchronize(); 

    cutStopTimer(cuProfile->kernelTime);
    
    cuProfile->kernelDuration = cutGetTimerValue(cuProfile->kernelTime);
    cuProfile->kernelExecErr = hipPeekAtLastError(); 
}

extern 
void cuFFTShift_2D_Double_Complex( dim3 cuBlock, dim3 cuGrid, 
                                   hipfftDoubleComplex* devArrayOutput, hipfftDoubleComplex* devArrayInput, 
                                   int nX,
                                   cudaProfile* cuProfile)
{
    cutCreateTimer(&(cuProfile->kernelTime));
    cutResetTimer(cuProfile->kernelTime);
    cutStartTimer(cuProfile->kernelTime);
    
    fftShift_2D_Double_Complex_Kernel <<< cuGrid, cuBlock>>> (devArrayOutput, devArrayInput, nX); 
    hipDeviceSynchronize(); 

    cutStopTimer(cuProfile->kernelTime);
    
    cuProfile->kernelDuration = cutGetTimerValue(cuProfile->kernelTime);
    cuProfile->kernelExecErr = hipPeekAtLastError(); 
}

extern 
void cuFFTShift_3D( dim3 cuBlock, dim3 cuGrid, 
                    float* devArrayOutput, float* devArrayInput, 
                    int nX, 
                    cudaProfile* cuProfile)
{
    cutCreateTimer(&(cuProfile->kernelTime));
    cutResetTimer(cuProfile->kernelTime);
    cutStartTimer(cuProfile->kernelTime);
    
    fftShift_3D_i(devArrayInput, devArrayOutput, nX, cuBlock, cuGrid); 
    hipDeviceSynchronize(); 

    cutStopTimer(cuProfile->kernelTime);
    
    cuProfile->kernelDuration = cutGetTimerValue(cuProfile->kernelTime);
    cuProfile->kernelExecErr = hipPeekAtLastError(); 
}

extern 
void cuFFTShift_3D_Double( dim3 cuBlock, dim3 cuGrid, 
                           double* devArrayOutput, double* devArrayInput, 
                           int nX, 
                           cudaProfile* cuProfile)
{
    cutCreateTimer(&(cuProfile->kernelTime));
    cutResetTimer(cuProfile->kernelTime);
    cutStartTimer(cuProfile->kernelTime);
    
    fftShift_3D_Double_i(devArrayInput, devArrayOutput, nX, cuBlock, cuGrid);
    hipDeviceSynchronize(); 

    cutStopTimer(cuProfile->kernelTime);
    
    cuProfile->kernelDuration = cutGetTimerValue(cuProfile->kernelTime);
    cuProfile->kernelExecErr = hipPeekAtLastError();  
}

extern 
void cuFFTShift_3D_Complex( dim3 cuBlock, dim3 cuGrid, hipfftComplex* devArrayOutput, hipfftComplex* devArrayInput, 
                            int nX, 
                            cudaProfile* cuProfile)
{
    cutCreateTimer(&(cuProfile->kernelTime));
    cutResetTimer(cuProfile->kernelTime);
    cutStartTimer(cuProfile->kernelTime);
    
    fftShift_3D_Complex_i(devArrayInput, devArrayOutput, nX, cuBlock, cuGrid); 
    hipDeviceSynchronize(); 

    cutStopTimer(cuProfile->kernelTime);
    
    cuProfile->kernelDuration = cutGetTimerValue(cuProfile->kernelTime);
    cuProfile->kernelExecErr = hipPeekAtLastError(); 
}

extern 
void cuFFTShift_3D_Double_Complex( dim3 cuBlock, dim3 cuGrid, hipfftDoubleComplex* devArrayOutput, hipfftDoubleComplex* devArrayInput, 
                                   int nX, 
                                   cudaProfile* cuProfile)
{
    cutCreateTimer(&(cuProfile->kernelTime));
    cutResetTimer(cuProfile->kernelTime);
    cutStartTimer(cuProfile->kernelTime);
    
    fftShift_3D_Double_Complex_i(devArrayInput, devArrayOutput, nX, cuBlock, cuGrid); 
    hipDeviceSynchronize(); 

    cutStopTimer(cuProfile->kernelTime);
    
    cuProfile->kernelDuration = cutGetTimerValue(cuProfile->kernelTime);
    cuProfile->kernelExecErr = hipPeekAtLastError(); 
}

#endif // CU_EXTERNS_CU_
